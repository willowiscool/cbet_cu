#include "hip/hip_runtime.h"
#include <cstdio>
#include <chrono>
#include "cbet.hpp"
#include "cbet.cuh"
#include "consts.hpp"
#include "utils.cuh"
#include "structs.hpp"

#define THREADS_PER_BLOCK 512

using namespace std::literals; // for dividing times by 1.0s

void cbet(MeshPoint* mesh, Crossing* crossings, RaystorePt* raystore) {
	/*printf("\tAllocating space for CBET, not counted in time\n");
	// TODO: make this use less memory by getting the minimum possible ncrossings value
	// (and maybe even the minimum possible nbeams value for the second nbeams in cbet_crosses?)
	double* coupling_mults = new double[consts::NBEAMS*consts::NRAYS*consts::NCROSSINGS*consts::NBEAMS]();
	double* w_mult_values = new double[consts::NBEAMS*consts::NRAYS*consts::NCROSSINGS]();*/

	auto start_time = std::chrono::high_resolution_clock::now();
	
	MeshPoint* cuda_mesh;
	Crossing* cuda_crossings;
	RaystorePt* cuda_raystore;
	double* cuda_w_mult_values;

	gpuErrchk(hipMalloc(&cuda_mesh, sizeof(MeshPoint) * consts::GRID));
	gpuErrchk(hipMemcpy(cuda_mesh, mesh, sizeof(MeshPoint) * consts::GRID, hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc(&cuda_crossings, sizeof(Crossing) * consts::NCROSSINGS * consts::NRAYS * consts::NBEAMS));
	gpuErrchk(hipMemcpy(cuda_crossings, crossings, sizeof(Crossing) * consts::NCROSSINGS * consts::NRAYS * consts::NBEAMS, hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc(&cuda_raystore, sizeof(RaystorePt) * consts::GRID * consts::NBEAMS));
	gpuErrchk(hipMemcpy(cuda_raystore, raystore, sizeof(RaystorePt) * consts::GRID * consts::NBEAMS, hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc(&cuda_w_mult_values, sizeof(double) * consts::NBEAMS * consts::NRAYS * consts::NCROSSINGS));
	gpuErrchk(hipMemset(cuda_w_mult_values, 0, sizeof(double) * consts::NBEAMS * consts::NRAYS * consts::NCROSSINGS));

	// fill remaining memory with coupling multipliers
	// another smart thing to do would be to have indices associated w/ each one
	// so we don't get a bunch of zeroes...
	size_t gpu_bytes_free;
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemGetInfo(&gpu_bytes_free, NULL));
	size_t num_cmults = gpu_bytes_free / sizeof(double);
	num_cmults -= 1<<19;
	double* cuda_coupling_mults;
	gpuErrchk(hipMalloc(&cuda_coupling_mults, sizeof(double) * num_cmults));
	gpuErrchk(hipMemset(cuda_coupling_mults, 0, sizeof(double) * num_cmults));
	size_t total_cmults = consts::NBEAMS*consts::NRAYS*consts::NCROSSINGS*consts::NRAYS;
	printf("\tCalculating and saving %lu/%lu coupling mults (%lf%%)\n",
			num_cmults, total_cmults, (double)num_cmults/(double)total_cmults);
	calc_coupling_mults
		<<<CEIL_DIV(num_cmults / (consts::NBEAMS*consts::NCROSSINGS), THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>
		(cuda_mesh, cuda_crossings, cuda_raystore, cuda_coupling_mults, num_cmults);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	printf("\tRunning CBET loop\n");
	double updateconv;
	double* cuda_updateconv;
	gpuErrchk(hipMalloc(&cuda_updateconv, sizeof(double)));
	gpuErrchk(hipMemset(cuda_updateconv, 0, sizeof(double)));

	double currmax = consts::MAX_INCR;
	size_t i;
	for (i = 1; i <= 500; i++) {
		printf("\t\tIteration %lu\n", i);
		get_cbet_gain
			<<<CEIL_DIV(consts::NRAYS * consts::NBEAMS, THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>
			(cuda_mesh, cuda_crossings, cuda_raystore, cuda_w_mult_values,
			 cuda_coupling_mults, num_cmults);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		gpuErrchk(hipMemset(cuda_updateconv, 0, sizeof(double)));
		update_intensities
			<<<CEIL_DIV(consts::NRAYS * consts::NBEAMS, THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>
			(cuda_crossings, cuda_raystore,
			cuda_w_mult_values, currmax, cuda_updateconv);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		gpuErrchk(hipMemcpy(&updateconv, cuda_updateconv, sizeof(double), hipMemcpyDeviceToHost));
		printf("\t\tupdateconv: %lf\n", updateconv);
		if (updateconv <= consts::CONVERGE) break;
		double currmaxa = consts::MAX_INCR*pow(consts::CBETCONVERGENCE, i);
		double currmaxb = consts::CBETCONVERGENCE*updateconv;
		currmax = fmin(currmaxa, currmaxb);
	}
	printf("\tIterated %lu times, running post\n", i-1);

	gpuErrchk(hipMemcpy(crossings, cuda_crossings, sizeof(Crossing) * consts::NCROSSINGS * consts::NRAYS * consts::NBEAMS, hipMemcpyDeviceToHost));
	post(mesh, crossings);

	gpuErrchk(hipFree(cuda_updateconv));
	gpuErrchk(hipFree(cuda_mesh));
	gpuErrchk(hipFree(cuda_crossings));
	gpuErrchk(hipFree(cuda_raystore));
	gpuErrchk(hipFree(cuda_w_mult_values));

	auto end_time = std::chrono::high_resolution_clock::now();
	printf("\tTotal time: %Lf seconds\n", (end_time - start_time) / 1.0s);
}

__global__ void get_cbet_gain(MeshPoint* mesh, Crossing* crossings, RaystorePt* raystore,
		double* w_mult_values, double* coupling_mults, size_t num_cmults) {
	size_t thread_num = blockIdx.x*THREADS_PER_BLOCK + threadIdx.x;
	if (thread_num > consts::NBEAMS * consts::NRAYS) return;

	size_t ind_offset = (thread_num)*consts::NCROSSINGS;
	Crossing* cross = crossings + ind_offset;
	double* w_mult = w_mult_values + ind_offset;
	ind_offset *= consts::NBEAMS; // from now on stores cmult index
	while (cross->i_b != 0) {
		double cbet_sum = 0.0;
		for (size_t o_b_num = 0; o_b_num < consts::NBEAMS; o_b_num++) {
			if (o_b_num == thread_num / consts::NRAYS) continue;
			RaystorePt* pt = get_pt(raystore + consts::GRID * o_b_num, cross->boxes);
			if (pt->cnum == 0 && pt->raynum == 0) continue;
			Crossing* raycross = crossings + (o_b_num*consts::NRAYS + pt->raynum)*consts::NCROSSINGS + pt->cnum;
			double avg_intensity = (raycross+1)->i_b > 0 ?
				(raycross->i_b+(raycross+1)->i_b)/2 :
				raycross->i_b;
			double coupling_mult = (ind_offset + o_b_num) > num_cmults ?
				get_coupling_mult(mesh, cross, raycross) :
				coupling_mults[ind_offset + o_b_num];
			cbet_sum += coupling_mult * avg_intensity;
		}
		*w_mult = exp(-1.0*cbet_sum) * cross->absorb_coeff;

		ind_offset += consts::NBEAMS;
		cross++;
		w_mult++;
	}
}

__global__ void update_intensities(Crossing* crossings, RaystorePt* raystore,
		double* w_mult_values, double curr_max, double* updateconv) {
	size_t thread_num = blockIdx.x*THREADS_PER_BLOCK + threadIdx.x;
	if (thread_num > consts::NBEAMS * consts::NRAYS) return;

	double conv_max = 0.0;
	size_t ind_offset = (thread_num)*consts::NCROSSINGS;
	Crossing* cross = crossings + ind_offset;
	double* lcl_w_mult = w_mult_values + ind_offset;
	double i0 = cross->i_b;
	double mult_acc = 1.0;
	size_t cnum = 1;
	lcl_w_mult++;
	cross++;
	while (cross->i_b != 0) {
		double new_intensity = limit_energy(cross->i_b, i0, mult_acc, curr_max, &conv_max);
		mult_acc *= *lcl_w_mult;
		cross->i_b = new_intensity;
		cross++;
		lcl_w_mult++;
		cnum++;
	}
	if (conv_max > *updateconv) *updateconv = conv_max;
}

__device__ double limit_energy(double i_prev, double i0, double mult_acc, double curr_max, double* max_change) {
	double i_curr = i0*mult_acc;
	// the fractional change in energy from imposing the update as is
	double fractional_change = abs(i_curr-i_prev)/i_prev;
	// update the convergence check variable
	*max_change = fmax(fractional_change, *max_change);
	// if the fractional change is too large, clamp the value
	if (fractional_change > curr_max) {
		int sign = (i_curr - i_prev > 0) ? 1 : -1;
		double correction = 1 + curr_max*sign;
		i_curr = i_prev*correction;
	}
	return i_curr;
}

// for all coupling multipliers
// UNUSED!!
__global__ void calc_coupling_mults(MeshPoint* mesh, Crossing* crossings, RaystorePt* raystore, double* coupling_mults, size_t num_cmults) {
	size_t thread_num = blockIdx.x*THREADS_PER_BLOCK + threadIdx.x;
	size_t ind_offset = thread_num*consts::NCROSSINGS;
	if (ind_offset*consts::NBEAMS >= num_cmults) return;

	Crossing* cross = crossings + ind_offset;
	ind_offset *= consts::NBEAMS;
	if (ind_offset >= num_cmults) return;
	size_t cnum = 0;
	while (cross->i_b != 0) {
		for (size_t o_b_num = 0; o_b_num < consts::NBEAMS; o_b_num++) {
			if (o_b_num == thread_num / consts::NRAYS) continue;
			if (ind_offset + o_b_num >= num_cmults) return;
			RaystorePt* pt = get_pt(raystore + consts::GRID * o_b_num, cross->boxes);
			if (pt->cnum == 0 && pt->raynum == 0) continue;
			coupling_mults[ind_offset + o_b_num] = get_coupling_mult(
				mesh, cross,
				crossings + (o_b_num*consts::NRAYS + pt->raynum)*consts::NCROSSINGS + pt->cnum);
		}

		cross++;
		cnum++;
		ind_offset += consts::NBEAMS;
	}
}

// for one coupling multiplier
__device__ double get_coupling_mult(MeshPoint* mesh, Crossing* cross, Crossing* raycross) {
	// copied in wholesale from Shuang
	MeshPoint* mesh_pt = get_pt(mesh, cross->boxes);
	double area_avg = (raycross+1)->i_b != 0 ?
		(raycross->area_ratio+(raycross+1)->area_ratio)/2.0 :
		raycross->area_ratio;
	double ne_over_nc = mesh_pt->eden;
	if (ne_over_nc > consts::NCRIT) ne_over_nc = 0.99;
	else ne_over_nc = ne_over_nc / consts::NCRIT;
	double ne_over_nc_corrected = fmin(ne_over_nc, 1.0); // TODO can remove?
	double ne_term = sqrt(1 - ne_over_nc_corrected);
	double epsilon_eff = ne_term * ne_term;
	double interaction_mult = 1/(area_avg*ne_term)*1/sqrt(epsilon_eff);

	Xyz<double> k_seed = cross->dk;
	Xyz<double> k_pump = raycross->dk;

	double omega1 = consts::OMEGA, omega2 = consts::OMEGA;

	Xyz<double> iaw_vector = {
		(omega1*k_seed.x - omega2*k_pump.x)*sqrt(1-ne_over_nc)/consts::C_SPEED,
		(omega1*k_seed.y - omega2*k_pump.y)*sqrt(1-ne_over_nc)/consts::C_SPEED,
		(omega1*k_seed.z - omega2*k_pump.z)*sqrt(1-ne_over_nc)/consts::C_SPEED
	};
	double k_iaw = mag(iaw_vector);
	double eta_numerator = omega1-omega2 -
		(iaw_vector.x * mesh_pt->machnum.x +
		 iaw_vector.y * mesh_pt->machnum.y + 
		 iaw_vector.z * mesh_pt->machnum.z); // * consts::CS
	double eta_denominator = k_iaw; // * consts::CS
	double eta = eta_numerator/eta_denominator;

	// THIS ONE IS CONSTANT TODO MOVE TO CONSTS...
	double param1 = consts::CBET_CONST / (consts::OMEGA*(consts::TE_EV/1e3 + 3.0 * consts::TI_EV/1e3/consts::Z));
	double param2 = ne_over_nc/consts::IAW*consts::IAW*consts::IAW*eta;
	double param3 = pow(eta*eta-1.0, 2) + consts::IAW*consts::IAW*eta*eta;
	double param4 = interaction_mult;

	double coupling_mult = param1*param2/param3*param4*cross->dkmag;
	// Random polarization
	coupling_mult *= (1 + pow(k_seed.x * k_pump.x + k_seed.y * k_pump.y + k_seed.z * k_pump.z, 2)) / 4;
	
	return coupling_mult;
}

void post(MeshPoint* mesh, Crossing* crossings) {
	const double norm_factor_const = sqrt(8.0*M_PI/consts::C_SPEED) * consts::ESTAT / (consts::ME_G * consts::C_SPEED * consts::W0) * sqrt(1e14 * 1e7);
	for (size_t beamnum = 0; beamnum < consts::NBEAMS; beamnum++) {
		for (size_t raynum = 0; raynum < consts::NRAYS; raynum++) {
			Crossing* cross = crossings + ((beamnum*consts::NRAYS) + raynum) * consts::NCROSSINGS;
			size_t cnum = 0;
			while (cross->i_b != 0) {
				double area_avg = (cross+1)->i_b != 0 ?
					(cross->area_ratio+(cross+1)->area_ratio)/2.0 :
					cross->area_ratio;
				double ne_over_nc = get_pt(mesh, cross->boxes)->eden;
				if (ne_over_nc > consts::NCRIT) ne_over_nc = 9.04e21; // hard coded??
				ne_over_nc = ne_over_nc / consts::NCRIT;
				double ne_over_nc_corrected = fmin(ne_over_nc, 1.0); // TODO can remove?
				double ne_term = sqrt(1 - ne_over_nc_corrected);
				double epsilon_eff = ne_term * ne_term;
				double interaction_mult = 1/(area_avg*ne_term)*1/sqrt(epsilon_eff);
				double norm_factor = norm_factor_const * sqrt(interaction_mult) * pow(epsilon_eff, 0.25);
				double prev_intensity = cross->i_b;
				cross->i_b = sqrt(prev_intensity) * norm_factor;

				double absorption = cnum < 2 ? 0 : (1 - cross->kds / (cross-1)->kds);
				// todo: why?
				double power = prev_intensity * 1e2 * pow((consts::BEAM_MAX_Z-consts::BEAM_MIN_Z) / (consts::NRAYS_X-1), 2.0);
				cross->absorption_data = absorption * power;

				cross++;
				cnum++; // used in absorption
			}
		}
	}
}
